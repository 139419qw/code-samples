/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "common.h"

#define TILE_DIM 32
#define BLOCK_ROWS 8

////////////////////////////////////////////////////////////////////////////////
// Callback Implementations
////////////////////////////////////////////////////////////////////////////////
__device__ hipfftReal CB_ConvertInputR(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
    char element = ((char*)dataIn)[offset];
    return (hipfftReal)((float)element/127.0f);
}

__device__ hipfftCallbackLoadR d_loadCallbackPtr = CB_ConvertInputR; 

__device__ void CB_ConvolveAndStoreTransposedC(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
    hipfftComplex *filter = (hipfftComplex*)callerInfo;
    size_t row = offset / COMPLEX_SIGNAL_SIZE;
    size_t col = offset % COMPLEX_SIGNAL_SIZE;

    ((hipfftComplex*)dataOut)[col * BATCH_SIZE + row] = ComplexMul(element, filter[col]);
}

__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_ConvolveAndStoreTransposedC;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, const char **argv)
{
    struct hipDeviceProp_t properties;
    int device = argc > 1 ? atoi(argv[1]) : 0;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&properties, device));
    if( !(properties.major >= 2) ) {
        printf("This sample requires CUDA architecture SM2.0 or higher\n");
        exit(EXIT_FAILURE);
    }

    // Allocate and initialize memory
    printf("Preparing input: %dx%d\n", BATCH_SIZE, INPUT_SIGNAL_SIZE);
    char *_8bit_signal;
    hipfftComplex *result, *filter;

    checkCudaErrors(hipMallocManaged(&_8bit_signal, sizeof(char) * INPUT_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&result, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&filter, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE, hipMemAttachGlobal));

    initInputs(_8bit_signal, filter);
    
    //compute reference result for later verification
    printf("Computing reference solution\n");
    hipfftComplex *reference = computeReference(_8bit_signal, filter);

    printf("Creating FFT plan\n");
    hipfftHandle fftPlan;
    size_t workSize;
    
    checkCudaErrors(hipfftCreate(&fftPlan));
    int signalSize = INPUT_SIGNAL_SIZE;
    checkCudaErrors(hipfftMakePlanMany(fftPlan, 1, &signalSize, 0,0,0,0,0,0, HIPFFT_R2C, BATCH_SIZE, &workSize));

    /*
     * Retrieve address of callback functions on the device
     */                              
    hipfftCallbackLoadR h_loadCallbackPtr;
    hipfftCallbackStoreC h_storeCallbackPtr;
    checkCudaErrors(hipMemcpyFromSymbol(&h_loadCallbackPtr, 
                                          HIP_SYMBOL(d_loadCallbackPtr), 
                                          sizeof(h_loadCallbackPtr)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr, 
                                          HIP_SYMBOL(d_storeCallbackPtr), 
                                          sizeof(h_storeCallbackPtr)));

    // Now associate the callbacks with the plan.
    hipfftResult status = hipfftXtSetCallback(fftPlan, 
                            (void **)&h_loadCallbackPtr, 
                            HIPFFT_CB_LD_REAL,
                            0);
    if (status == CUFFT_LICENSE_ERROR) {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
    } else {
        checkCudaErrors(status);
    }

   checkCudaErrors(hipfftXtSetCallback(fftPlan, 
                                (void **)&h_storeCallbackPtr, 
                                HIPFFT_CB_ST_COMPLEX,
                                (void **)&filter));

    //create timers
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float elapsedTime;

    printf("Running %d iterations\n", ITERATIONS);
    checkCudaErrors(hipEventRecord(start, 0));

    /*
     * The actual Computation
     */

    for(int i = 0; i < ITERATIONS; i++) {
        checkCudaErrors(hipfftExecR2C(fftPlan, (hipfftReal*)_8bit_signal, result));
    }

    checkCudaErrors(hipEventRecord(end, 0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    printf("Time for the FFT: %fms\n", elapsedTime);

    //Verify correct result    
    if(postprocess(reference, result, COMPLEX_SIGNAL_SIZE * BATCH_SIZE)) {
        printf("Verification successful.\n");
    } else {
        printf("!!! Verification Failed !!!\n");
    }

    //Cleanup
    checkCudaErrors(hipfftDestroy(fftPlan));

    checkCudaErrors(hipFree(_8bit_signal));
    checkCudaErrors(hipFree(result));
    checkCudaErrors(hipFree(filter));
    checkCudaErrors(hipFree(reference));
      
    //clean up driver state
    hipDeviceReset();

    printf("Done\n");
    
    return 0;
}